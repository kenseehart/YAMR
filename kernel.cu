#include "hip/hip_runtime.h"
﻿/*
MIT License

Copyright(c) 2021 kenseehart

Permission is hereby granted, free of charge, to any person obtaining a copy of
this softwareand associated documentation files(the "Software"), to deal in the
Software without restriction, including without limitation the rights to use,
copy, modify, merge, publish, distribute, sublicense, and /or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions :

The above copyright noticeand this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

const float float_min = -3.402e+38;

__device__ const float d_float_min = -3.402e+38;

__host__ void _cudaCheckError(const char* file, int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Fatal error: %s at %s[%d]\n", hipGetErrorString(err), file, line);
        fprintf(stderr, "*** CUDA FAILED - ABORTING\n");
        exit(1);
    }
}

#define cudaCheckError _cudaCheckError(__FILE__, __LINE__)


__global__ void maxReduce(volatile float* d_data)
{
    // compute max over all threads, store max in d_data[0]
    int i = threadIdx.x;
    __shared__ volatile float max_value;

    if (i == 0) max_value = d_float_min;

    float v = d_data[i];
    __syncthreads();

    while (max_value < v)
    {
        max_value = v;
    }

    __syncthreads();
    if (i == 0) d_data[0] = max_value;
}


void testMax(int n, bool verbose)
{
    float* h_data, * d_data;
    float cpu_max = float_min;

    // Allocate memory mapped data
    hipHostAlloc((void**)&h_data, n * sizeof(float), hipHostMallocMapped); cudaCheckError;
    hipHostGetDevicePointer((int**)&d_data, (int*)h_data, 0); cudaCheckError;

    for (int i = 0; i < n; i++)
    {
        // randomize
        h_data[i] = (float)rand() / (float)(1 + rand());

        // get cpu opinion of the max for testing
        if (cpu_max < h_data[i]) cpu_max = h_data[i];
    }

    // run the kernel
    maxReduce << <1, n >> > (d_data); cudaCheckError;
    hipDeviceSynchronize(); cudaCheckError;

    // did the gpu get the same answer as the cpu?
    if (verbose)
    {
        printf("n =%6d cpu_max =%12.4f, gpu_max =%12.4f, result = %s\n", n, cpu_max, h_data[0], (cpu_max == h_data[0]) ? "PASS" : "FAIL");
    }
    else
    {
        if (cpu_max != h_data[0])
        printf("FAIL");
    }
}


int main()
{
    for (int j = 1; j < 5000; j++)
    {
        for (int i = 1; i < 16; i++)
        {
            testMax(i*31, j<20);
        }
    }
    return 0;
}
