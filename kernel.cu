#include "hip/hip_runtime.h"
﻿/*
MIT License

Copyright(c) 2021 kenseehart

Permission is hereby granted, free of charge, to any person obtaining a copy of
this softwareand associated documentation files(the "Software"), to deal in the
Software without restriction, including without limitation the rights to use,
copy, modify, merge, publish, distribute, sublicense, and /or sell copies of
the Software, and to permit persons to whom the Software is furnished to do so,
subject to the following conditions :

The above copyright noticeand this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A
PARTICULAR PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT
HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__device__ const float float_min = -3.402e+38;

__host__ void _cudaCheckError(const char* file, int line)
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Fatal error: %s at %s[%d]\n", hipGetErrorString(err), file, line);
        fprintf(stderr, "*** CUDA FAILED - ABORTING\n");
        exit(1);
    }
}

#define cudaCheckError _cudaCheckError(__FILE__, __LINE__)


__global__ void maxReduce(float* d_data)
{
    // compute max over all threads, store max in d_data[0]
    int i = threadIdx.x;
    __shared__ float max_value;

    if (i == 0) max_value = float_min;
    __syncthreads();

    float v = d_data[i];

    while (max_value < v)
    {
        max_value = v;
    }

    __syncthreads();
    if (i == 0) d_data[0] = max_value;
}


void testMax(int n)
{
    float* h_data, * d_data;
    float cpu_max = float_min;

    // Allocate memory mapped data
    hipHostAlloc((void**)&h_data, n * sizeof(float), hipHostMallocMapped); cudaCheckError;
    hipHostGetDevicePointer((int**)&d_data, (int*)h_data, 0); cudaCheckError;

    for (int i = 0; i < n; i++)
    {
        // randomize
        h_data[i] = -(float)rand() / (float)(1 + rand());

        // get cpu opinion of the max for testing
        if (cpu_max < h_data[i]) cpu_max = h_data[i];
    }

    // run the kernel
    maxReduce << <1, n >> > (d_data); cudaCheckError;
    hipDeviceSynchronize(); cudaCheckError;

    // did the gpu get the same answer as the cpu?
    printf("cpu_max = %f, gpu_max = %f, result = %s\n", cpu_max, h_data[0], (cpu_max == h_data[0]) ? "PASS" : "FAIL");
}


int main()
{

    for (int i = 1; i < 150; i++)
    {
        testMax(i * 5);
    }

    return 0;
}
